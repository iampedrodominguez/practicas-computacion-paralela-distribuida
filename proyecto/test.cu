
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define INF INT_MAX
using namespace std;

#define N 10
typedef vector<vector<int>> Graph;

__device__ int cost;

__global__ void test(int graph[], int n)

{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < n) {
        int mn = INF;

        //Find the smallest row value
        for (int j = 0; j < n; j++)
            if (graph[idx * n + j] < mn)
                mn = graph[idx * n + j];
        
        //If small value is valid
        if (mn != INF && mn > 0)
        {   
            cost += mn;

            //Reduce the cost for each row element
            for(int j=0; j<n; j++){
                if(graph[idx * n + j] != INF){
                    graph[idx * n + j] -= mn;
                }
            }
        }
    }
}

int main(){

    int* cost;
    const int n = 10;

    //Allocatig GPU memory for matrix
   
    int* CPUgraph = new int[n * n];
    int* GPUgraph = new int[n * n];


    for (int i = 0 ; i < n; i++) {
        for (int j = 0; j < n; j++)
            CPUgraph[i * n + j] = 10;
    }

    size_t bytes_i = n * n * sizeof(int);

    hipMalloc((void**)&GPUgraph, bytes_i);

    //Allocating GPU memory for global variable COST
    hipMalloc((void**)&cost, sizeof(int));

    //Pensar como pasarlo a GPU
    hipMemcpy(GPUgraph, CPUgraph, bytes_i, hipMemcpyHostToDevice); 

    //Declare dimensions of GPU grid
    dim3 BLOCKS(n, n);

    //Call reduce functions in GPU
    test<<<1, BLOCKS>>>(GPUgraph, n);

    //cout << GPUgraph[0];
    //Return matrix
    if (hipMemcpy(CPUgraph, GPUgraph, bytes_i, hipMemcpyDeviceToHost) == hipSuccess){
        cout << "Se pudo copiar a matriz final\n";
    } 


    for (int i = 0 ; i < n; i++) {
        for (int j = 0; j < n; j++)
            cout << CPUgraph[i * n + j] << " ";
        cout << endl;
    }
    //Free memory from GPU
    hipFree(GPUgraph);
    hipFree(cost);

    return 0;
}