#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#define INF INT_MAX
using namespace std;

typedef vector<vector<int>> Graph;

__device__ int cost;

__global__ void test(Graph graph, int n)

{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < n) {
        int mn = INF;

        //Find the smallest row value
        for (int j = 0; j < n; j++)
            if (graph[idx][j] < mn)
                mn = graph[idx][j];
        
        //If small value is valid
        if (mn != INF && mn > 0)
        {   
            cost += mn;

            //Reduce the cost for each row element
            for(int j=0; j<n; j++){
                if(graph[idx][j] != INF){
                    graph[idx][j] -= mn;
                }
            }
        }
    }
}

int main(){

    int* cost;
    int n = 10;

    //Allocatig GPU memory for each matrix
    Graph CPUgraph = vector<vector<int>>(vector<int>(10),10);
    Graph* GPUgraph = new Graph(10,10,0);

    size_t bytes_i = n * n * sizeof(int);
    hipMalloc((void**)GPUgraph, bytes_i);

    //Allocating GPU memory for global variable cost
    hipMalloc(&cost, sizeof(int));

    //Pensar como pasarlo a GPU
    hipMemcpy(GPUgraph, &(CPUgraph), bytes_i, hipMemcpyHostToDevice); 

    //Declare dimensions of GPU grid
    dim3 BLOCKS(n, n);

    //Call reduce functions in GPU
    test<<<1, BLOCKS>>>(*GPUgraph, n);

    //Return matrix
    if (hipMemcpy(&(CPUgraph), GPUgraph, bytes_i, hipMemcpyDeviceToHost) == hipSuccess){
        printf("Se pudo copiar a imagen final\n");
    } 

    for (auto row : CPUgraph) {
       for (auto col : row)
              cout << col << " ";
       cout << endl;
    }

    //Free memory from GPU
    hipFree(GPUgraph);
    hipFree(cost);

    return 0;
}