
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define INF INT_MAX
using namespace std;

#define N 10
typedef vector<vector<int>> Graph;

__global__ void reduceGraphRows(int graph[], int n, int cost[])

{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //int idy = threadIdx.y + blockIdx.y * blockDim.y;

    
    if (idx < n) {
        int mn = INF;

        //Find the smallest row value
        for (int j = 0; j < n; j++)
            if (graph[idx * n + j] < mn)
                mn = graph[idx * n + j];
        
        //If small value is valid
        if (mn != INF && mn > 0)
        {   
            
            cost[idx] = mn;

            //Reduce the cost for each row element
            for(int j=0; j<n; j++){
                if(graph[idx * n + j] != INF){
                    graph[idx * n + j] -= mn;
                }
            }
        }
    }
}


__global__ void reduceGraphColumns(int graph[], int n, int cost[])
{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n)
    {
        int mn = INF;
        for (int j=0; j<n; j++)
            if (graph[j * n + idx] < mn)
                mn = graph[j * n + idx];

        if(mn != INF && mn > 0)
        {    
            cost[idx] += mn;
            for(int j=0; j<n; j++)

                if(graph[j * n + idx] != INF)
                    graph[j * n + idx] -= mn;
        }
    }
}

int main(){

    const int n = 10;
    int* CPUcost = new int[n];
    int* GPUcost = new int[n];

    //Allocatig GPU memory for matrix
   
    int* CPUgraph = new int[n * n];
    int* GPUgraph = new int[n * n];


    for (int i = 0 ; i < n; i++) {
        CPUcost[i] = 0;
        for (int j = 0; j < n; j++)
            CPUgraph[i * n + j] = 10;
    }

    size_t bytes_m = n * n * sizeof(int);
    size_t bytes_c = n * sizeof(int);

    hipMalloc((void**)&GPUgraph, bytes_m);

    //Allocating GPU memory for global variable COST
    hipMalloc((void**)&GPUcost, bytes_c);

    //Sending data to GPU
    hipMemcpy(GPUgraph, CPUgraph, bytes_m, hipMemcpyHostToDevice); 
    hipMemcpy(GPUcost, CPUcost, bytes_c, hipMemcpyHostToDevice); 

    //Declare dimensions of GPU grid
    dim3 BLOCKS(n, n);

    //Call reduce functions in GPU
    reduceGraphRows<<<1, BLOCKS>>>(GPUgraph, n, GPUcost);
    reduceGraphColumns<<<1, BLOCKS>>>(GPUgraph, n, GPUcost);

    //Return matrix
    hipMemcpy(CPUgraph, GPUgraph, bytes_m, hipMemcpyDeviceToHost);
    hipMemcpy(CPUcost, GPUcost, bytes_c, hipMemcpyDeviceToHost);

    int globalCost = 0;

    for (int i = 0 ; i < n; i++) {
        globalCost += CPUcost[i];
        for (int j = 0; j < n; j++)
            cout << CPUgraph[i * n + j] << " ";
        cout << endl;
    }

    
    cout << globalCost;
    //Free memory from GPU
    hipFree(GPUgraph);
    hipFree(GPUcost);

    return 0;
}