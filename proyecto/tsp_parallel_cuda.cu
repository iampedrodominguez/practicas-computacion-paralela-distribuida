
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stack>
#include <queue>
#include <climits>
#include <algorithm>
#include <math.h>
#include <fstream>
#include <omp.h>

using namespace std;

#define INF INT_MAX

typedef vector<vector<int>> Graph;

class Node {
public:
    Graph graph;
    int cost;
    int cost_parent;

    Node() {
        cost = 0;
        cost_parent = 0;
    }
};

class TSP

{
private:
    int n;
    Graph graph;
    Node* root;
    vector<int> path;
    int p;
    void reduceGraph(Node* node);
    void reduce(Node* node, int from, int to);

public:
    TSP(int n, Graph graph, int p);
    void printGraph();
    void solve();
    bool printSolution();
    bool printSolutionToFile(string filename);
    int getScore();
};

TSP::TSP(int n, Graph graph, int p)
{
    this->n = n;
    this->graph = graph;
    this->p = p;
    
    root = new Node;
    root->graph = graph;
    root->cost = 0;
    root->cost_parent = 0;
};


__global__ void reduceGraphColumns(int graph[], int n, int cost[])
{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n)
    {
        int mn = INF;
        for (int j=0; j<n; j++)
            if (graph[j * n + idx] < mn)
                mn = graph[j * n + idx];

        if(mn != INF && mn > 0)
        {    
            cost[idx] += mn;
            for(int j=0; j<n; j++)

                if(graph[j * n + idx] != INF)
                    graph[j * n + idx] -= mn;
        }
    }
}

__global__ void reduceGraphRows(int graph[], int n, int cost[])

{
    //Get thread ID.x
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //int idy = threadIdx.y + blockIdx.y * blockDim.y;

    
    if (idx < n) {
        int mn = INF;

        //Find the smallest row value
        for (int j = 0; j < n; j++)
            if (graph[idx * n + j] < mn)
                mn = graph[idx * n + j];
        
        //If small value is valid
        if (mn != INF && mn > 0)
        {   
            
            cost[idx] = mn;

            //Reduce the cost for each row element
            for(int j=0; j<n; j++){
                if(graph[idx * n + j] != INF){
                    graph[idx * n + j] -= mn;
                }
            }
        }
    }
}

void TSP::reduceGraph(Node* node)
{
    int* CPUcost = new int[n];
    int* GPUcost = new int[n];

    //Allocatig GPU memory for matrix
   
    int* CPUgraph = new int[n * n];
    int* GPUgraph = new int[n * n];

    for(int i = 0 ; i < n; i++) {
        CPUcost[i] = 0;
        for (int j = 0; j < n; j++)
            CPUgraph[i * n + j] = node->graph[i][j];
    }

    size_t bytes_m = n * n * sizeof(int);
    size_t bytes_c = n * sizeof(int);

    //Allocating GPU memory for global variable COST and GPUGRAPH
    hipMalloc((void**)&GPUgraph, bytes_m);
    hipMalloc((void**)&GPUcost, bytes_c);

    // Copy data to GPU
    hipMemcpy(GPUgraph, CPUgraph, bytes_m, hipMemcpyHostToDevice); 
    hipMemcpy(GPUcost, CPUcost, bytes_c, hipMemcpyHostToDevice); 

    // Declare dimensions of GPU grid
    dim3 BLOCKS(n, n);

    // Call reduce functions in GPU
    reduceGraphRows<<<1, BLOCKS>>>(GPUgraph, n, GPUcost);
    reduceGraphColumns<<<1, BLOCKS>>>(GPUgraph, n, GPUcost);

    // Copy data from GPU
    hipMemcpy(CPUgraph, GPUgraph, bytes_m, hipMemcpyDeviceToHost);
    hipMemcpy(CPUcost, GPUcost, bytes_c, hipMemcpyDeviceToHost);

    int globalCost = 0;

    for (int i = 0 ; i < n; i++) {
        globalCost += CPUcost[i];
        for (int j = 0; j < n; j++)
            cout << CPUgraph[i * n + j] << " ";
        cout << endl;
    }

    for(int i = 0 ; i < n; i++) {
        for (int j = 0; j < n; j++)
            node->graph[i][j] = CPUgraph[i * n + j];
    }
    
    node->cost += globalCost;

    // Free memory from GPU and CPU
    free(CPUgraph);
    free(CPUcost);
    hipFree(GPUgraph);
    hipFree(GPUcost);
}

void TSP::reduce(Node* node, int from, int to) {
    node->cost += node->graph[from][to] + node->cost_parent;
    for (int i = 0; i < n; i++) {
        node->graph[from][i] = INF;
        node->graph[i][to] = INF;
    }
    node->graph[to][0] = INF;
    reduceGraph(node);
}


void TSP::solve()
{
    path = {0};
    reduceGraph(root);

    auto cmp = [](pair<Node*, vector<int>> a, pair<Node*, vector<int>> b) {
        return a.first->cost > b.first->cost;
    };

    priority_queue<pair<Node*, vector<int>>, vector<pair<Node*, vector<int>>>, decltype(cmp)> pq(cmp);
    pq.push({root, {0}});

    while (!pq.empty()) {
        auto cur = pq.top();
        Node* curnode = cur.first;
        vector<int> curpath = cur.second;
        pq.pop();
        int from = curpath.back();
        if(curpath.size() == n)
        {
            curpath.push_back(0);
            path = curpath;
            break;
        }

#pragma omp parallel shared(pq, curnode, curpath)  num_threads(p)
{
        #pragma omp for 
        for(int to = 0; to < n; to++){
            if(curnode->graph[from][to] != INF)
            {
                //if to is elegible
                Node* child = new Node();
                child->graph = curnode->graph;
                vector<int> childpath = curpath;
                childpath.push_back(to);
                child->cost_parent = curnode->cost;

                reduce(child, from, to);
#pragma omp critical
                pq.push({child, childpath});
            }
        }
#pragma omp barrier
}

        delete curnode;
    }
} 

bool TSP::printSolution()
{
    if(path.size() == 0) {
        printf("Make solve to generate a solution\n");
        return 0;
    }
    int cost = 0;
    int prev = -1;
    printf("Solution: ");
    for (int &i : path) {
        printf("%d ", i);
        if (prev != -1) cost += graph[prev][i];
        prev = i;
    }
    printf("\nCost: %d\n", cost);
    return 1;
}

bool TSP::printSolutionToFile(string filename)
{
    if(!printSolution()) return 0;
    
    int cost = 0;
    int prev = -1;
    for (int &i : path) {
        if (prev != -1) cost += graph[prev][i];
        prev = i;
    }
    
    ofstream file;
    file.open(filename, ios::out);
    if(file.fail()) return 0;
    file << cost << "\n";
    for (int &i : path) {
        file << i + 1 << "\n";
    }
    file.close();
    return 1;
}

void TSP::printGraph()
{
    printf("Graph of size %d:\n", n);
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++)
            printf("%d ", graph[i][j]);
        printf("\n");
    }
    printf("\n");
}

int TSP::getScore()
{
    int cost = 0;
    int prev = -1;
    for (int &i : path) {
        if (prev != -1) cost += graph[prev][i];
        prev = i;
    }
    return cost;
}